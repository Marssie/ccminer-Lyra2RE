
extern "C"
{

#include "sph/neoscrypt.h"
#include "miner.h"
}

#include <stdint.h>

// aus cpu-miner.c
extern int device_map[8];

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void cuda_neoscrypt_cpu_init(int thr_id, int threads);
extern void cuda_neoscrypt_cpu_setBlock(void *pdata, const void *ptarget);
extern uint32_t cuda_neoscrypt_cpu_hash(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);


// X11 Hashfunktion
inline void neoscrypt_hash(void *state, const void *input)
{
    // blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11
	uint32_t hash[8];
	neoscrypt((uint8_t *)state, (uint8_t *)hash, 0x80000620);
    memcpy(state, hash, 32);
}


extern bool opt_benchmark;

extern "C" int scanhash_neoscrypt(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*256*8;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);

		cuda_neoscrypt_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	cuda_neoscrypt_cpu_setBlock((void*)endiandata,ptarget);
	

	do {
		int order = 0;


		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = cuda_neoscrypt_cpu_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			neoscrypt_hash(vhash64, endiandata);

			if ((vhash64[7]<=Htarg) && fulltest(vhash64, ptarget)) {

				pdata[19] = foundNonce;
				*hashes_done = foundNonce - first_nonce + 1;
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
