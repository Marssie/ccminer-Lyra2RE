#include "hip/hip_runtime.h"
/*
 * tiger-192 djm34
 * 
 */

/*
 * tiger-192 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 * 
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 * 
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>



#include "cuda_helper.h"


// aus heavy.cu

extern int device_major[8];

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);


__global__ void __launch_bounds__(512,2) m7_bigmul_unroll1_gpu(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{

//__shared__ uint64_t ams[38],bms[38],ws[38]
//ams
	                         int thread = (blockDim.x * blockIdx.x + threadIdx.x);

                             if (thread < threads)
                           {

#pragma unroll 32 
	for (int i=0;i<32;i++) {w[i*threads+thread]=0;}
#if __CUDA_ARCH__ < 500
#pragma unroll 32   
#endif
for (int i=0;i<32;i++) {
	uint64_t c=0;
	uint64_t u=0,v=0;  
    #pragma unroll 3
	for (int j=0;j<3;j++) {  
    muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);	
    w[(i+j)*threads+thread]=v;
    c=u; 
	}
   w[(i+3)*threads+thread]=u;
 }
							 } // threads
}

__global__ void __launch_bounds__(512,4) m7_bigmul_unroll1_gpu_50(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{

//__shared__ uint64_t ams[38],bms[38],ws[38]
//ams
	                         int thread = (blockDim.x * blockIdx.x + threadIdx.x);

                             if (thread < threads)
                           {

#pragma unroll 32 
	for (int i=0;i<32;i++) {w[i*threads+thread]=0;}
#if __CUDA_ARCH__ < 500
#pragma unroll 32   
#endif
for (int i=0;i<32;i++) {
	uint64_t c=0;
	uint64_t u=0,v=0;  
    #pragma unroll 3
	for (int j=0;j<3;j++) {  
    muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);	
    w[(i+j)*threads+thread]=v;
    c=u; 
	}
   w[(i+3)*threads+thread]=u;
 }
							 } // threads
}


__global__ void __launch_bounds__(512,2) m7_bigmul_unroll2_gpu(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{


	                         int thread = (blockDim.x * blockIdx.x + threadIdx.x);

                             if (thread < threads)
                           {


#if __CUDA_ARCH__ < 500
#pragma unroll
#endif
	for (int i=0;i<38;i++) {w[i*threads+thread]=0;}
#if __CUDA_ARCH__ < 500
#pragma unroll    
#endif
for (int i=0;i<35;i++) {
	uint64_t c=0;
	uint64_t u=0,v=0;  
#if __CUDA_ARCH__ < 500
    #pragma unroll
#endif
	for (int j=0;j<3;j++) {  
    muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);	
    w[(i+j)*threads+thread]=v;
    c=u; 
	}
   w[(i+3)*threads+thread]=u;
 }
//////////////////////////////////////////////////////////////////////////////////////////////////	  

 } //// threads
}

__global__ void __launch_bounds__(512,4) m7_bigmul_unroll2_gpu_50(int threads, uint64_t* am, uint64_t* bm, uint64_t *w)
{


	                         int thread = (blockDim.x * blockIdx.x + threadIdx.x);

                             if (thread < threads)
                           {


#if __CUDA_ARCH__ < 500
#pragma unroll
#endif
	for (int i=0;i<38;i++) {w[i*threads+thread]=0;}
#if __CUDA_ARCH__ < 500
#pragma unroll    
#endif
for (int i=0;i<35;i++) {
	uint64_t c=0;
	uint64_t u=0,v=0;  
#if __CUDA_ARCH__ < 500
    #pragma unroll
#endif
	for (int j=0;j<3;j++) {  
    muladd128(u,v,am[j*threads+thread],bm[i*threads+thread],w[(i+j)*threads+thread],c);	
    w[(i+j)*threads+thread]=v;
    c=u; 
	}
   w[(i+3)*threads+thread]=u;
 }
//////////////////////////////////////////////////////////////////////////////////////////////////	  

 } //// threads
}



__host__ void m7_bigmul_unroll1_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order)
{

	const int threadsperblock = 512; 

dim3 grid(threads/threadsperblock);
dim3 block(threadsperblock);
  
	size_t shared_size =0;
	if (device_major[thr_id]==5) {
		m7_bigmul_unroll1_gpu_50<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);}
	else {
		m7_bigmul_unroll1_gpu<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);}
}

__host__ void m7_bigmul_unroll2_cpu(int thr_id, int threads,uint64_t* Hash1, uint64_t* Hash2,uint64_t *finalHash,int order)
{

	const int threadsperblock = 512;

dim3 grid(threads/threadsperblock);
dim3 block(threadsperblock);
  
	size_t shared_size =0;

	if (device_major[thr_id]==5) {
		m7_bigmul_unroll2_gpu_50<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);}
	else {
		m7_bigmul_unroll2_gpu<<<grid, block, shared_size>>>(threads,Hash1,Hash2,finalHash);}

}


__host__ void m7_bigmul_init(int thr_id, int threads)
{
	// why I am here ?
}