#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>


extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
extern int compute_version[8];

#include "cuda_helper.h"


static __constant__ uint2 blake2b_IV[8] =
{
	{ 0xf3bcc908, 0x6a09e667  }, 
	{ 0x84caa73b, 0xbb67ae85  },
	{ 0xfe94f82b, 0x3c6ef372  },
	{ 0x5f1d36f1, 0xa54ff53a  },
	{ 0xade682d1, 0x510e527f  },
	{ 0x2b3e6c1f, 0x9b05688c  },
	{ 0xfb41bd6b, 0x1f83d9ab  },
	{ 0x137e2179, 0x5be0cd19  }
};
// data: 0-4 outputhash 4-8 outputhash 8-16 basil


#define reduceDuplexRowSetup(rowIn, rowInOut, rowOut) \
  { \
	for (int i = 0; i < 8; i++) \
			{ \
\
		for (int j = 0; j < 12; j++) {state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];} \
		round_lyra_v35(state); \
		for (int j = 0; j < 12; j++) {Matrix[j + 84 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];} \
\
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
			} \
 \
  } 

#define reduceDuplexRow(rowIn, rowInOut, rowOut) \
  { \
	 for (int i = 0; i < 8; i++) \
	 	 	 	 { \
		 for (int j = 0; j < 12; j++) \
			 state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
 \
		 round_lyra_v35(state); \
		 for (int j = 0; j < 12; j++) {Matrix[j + 12 * i][rowOut] ^= state[j];} \
\
		 Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		 Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		 Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		 Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		 Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		 Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		 Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		 Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		 Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		 Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		 Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		 Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
	 	 	 	 } \
 \
  } 
#define absorbblock(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
  } 
//// test version
#define reduceDuplexRowSetup_test(rowIn, rowInOut, rowOut) \
  { \
	for (int i = 0; i < 8; i++) \
				{ \
\
		for (int j = 0; j < 12; j++) {state[j] ^= Matrix[j][i][rowIn] + Matrix[j][i][rowInOut];} \
		round_lyra_v35(state); \
		for (int j = 0; j < 12; j++) {Matrix[j][7-i][rowOut] = Matrix[j][i][rowIn] ^ state[j];} \
\
		Matrix[0][i][rowInOut] ^= state[11]; \
		Matrix[1][i][rowInOut] ^= state[0]; \
		Matrix[2][i][rowInOut] ^= state[1]; \
		Matrix[3][i][rowInOut] ^= state[2]; \
		Matrix[4][i][rowInOut] ^= state[3]; \
		Matrix[5][i][rowInOut] ^= state[4]; \
		Matrix[6][i][rowInOut] ^= state[5]; \
		Matrix[7][i][rowInOut] ^= state[6]; \
		Matrix[8][i][rowInOut] ^= state[7]; \
		Matrix[9][i][rowInOut] ^= state[8]; \
		Matrix[10][i][rowInOut] ^= state[9]; \
		Matrix[11][i][rowInOut] ^= state[10]; \
				} \
 \
  } 

#define reduceDuplexRow_test(rowIn, rowInOut, rowOut) \
  { \
	 for (int i = 0; i < 8; i++) \
	 	 	 	 	 { \
		 for (int j = 0; j < 12; j++) \
			 state[j] ^= Matrix[j][i][rowIn] + Matrix[j][i][rowInOut]; \
 \
		 round_lyra_v35(state); \
		 for (int j = 0; j < 12; j++) {Matrix[j][i][rowOut] ^= state[j];} \
\
		 Matrix[0][i][rowInOut] ^= state[11]; \
		 Matrix[1][i][rowInOut] ^= state[0]; \
		 Matrix[2][i][rowInOut] ^= state[1]; \
		 Matrix[3][i][rowInOut] ^= state[2]; \
		 Matrix[4][i][rowInOut] ^= state[3]; \
		 Matrix[5][i][rowInOut] ^= state[4]; \
		 Matrix[6][i][rowInOut] ^= state[5]; \
		 Matrix[7][i][rowInOut] ^= state[6]; \
		 Matrix[8][i][rowInOut] ^= state[7]; \
		 Matrix[9][i][rowInOut] ^= state[8]; \
		 Matrix[10][i][rowInOut] ^= state[9]; \
		 Matrix[11][i][rowInOut] ^= state[10]; \
	 	 	 	 	 } \
 \
  } 
#define absorbblock_test(in)  { \
	state[0] ^= Matrix[0][0][ in]; \
	state[1] ^= Matrix[1][0][in]; \
	state[2] ^= Matrix[2][0][in]; \
	state[3] ^= Matrix[3][0][in]; \
	state[4] ^= Matrix[4][0][in]; \
	state[5] ^= Matrix[5][0][in]; \
	state[6] ^= Matrix[6][0][in]; \
	state[7] ^= Matrix[7][0][in]; \
	state[8] ^= Matrix[8][0][in]; \
	state[9] ^= Matrix[9][0][in]; \
	state[10] ^= Matrix[10][0][in]; \
	state[11] ^= Matrix[11][0][in]; \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
	round_lyra_v35(state); \
  } 


//// compute 30 version 
#define reduceDuplexRowSetup_v30(rowIn, rowInOut, rowOut) \
  { \
	for (int i = 0; i < 8; i++) \
				{ \
\
		for (int j = 0; j < 12; j++) {state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];} \
		round_lyra_v30(state); \
		for (int j = 0; j < 12; j++) {Matrix[j + 84 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];} \
\
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
				} \
 \
  } 

#define reduceDuplexRow_v30(rowIn, rowInOut, rowOut) \
  { \
	 for (int i = 0; i < 8; i++) \
	 	 	 	 	 { \
		 for (int j = 0; j < 12; j++) \
			 state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
 \
		 round_lyra_v30(state); \
		 for (int j = 0; j < 12; j++) {Matrix[j + 12 * i][rowOut] ^= state[j];} \
\
		 Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		 Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		 Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		 Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		 Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		 Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		 Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		 Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		 Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		 Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		 Matrix[10 + 12 * i][rowInOut] ^= state[9]; \
		 Matrix[11 + 12 * i][rowInOut] ^= state[10]; \
	 	 	 	 	 } \
 \
  } 
#define absorbblock_v30(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
	round_lyra_v30(state); \
  } 




 static __device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
 {
	 a += b; d ^= a; d = ROR2(d, 32);
	 c += d; b ^= c; b = ROR2(b, 24);
	 a += b; d ^= a; d = ROR2(d, 16);
	 c += d; b ^= c; b = ROR2(b, 63);
 }

 static __device__ __forceinline__ void Gfunc_v30(uint64_t & a, uint64_t &b, uint64_t &c, uint64_t &d)
 {
	 a += b; d ^= a; d = ROTR64(d, 32);
	 c += d; b ^= c; b = ROTR64(b, 24);
	 a += b; d ^= a; d = ROTR64(d, 16);
	 c += d; b ^= c; b = ROTR64(b, 63);
 }

#define round_lyra_v35_new(state) { \
 Gfunc_v35(state[0], state[4], state[8], state[12]); \
 Gfunc_v35(state[1], state[5], state[9], state[13]); \
 Gfunc_v35(state[2], state[6], state[10], state[14]); \
 Gfunc_v35(state[3], state[7], state[11], state[15]); \
 Gfunc_v35(state[0], state[5], state[10], state[15]); \
 Gfunc_v35(state[1], state[6], state[11], state[12]); \
 Gfunc_v35(state[2], state[7], state[8], state[13]); \
 Gfunc_v35(state[3], state[4], state[9], state[14]); \
}

static __device__ __forceinline__ void round_lyra_v35(uint2 *s) 
{
	Gfunc_v35(s[0], s[4], s[8],  s[12]);
	Gfunc_v35(s[1], s[5], s[9],  s[13]);
	Gfunc_v35(s[2], s[6], s[10], s[14]);
	Gfunc_v35(s[3], s[7], s[11], s[15]);
	Gfunc_v35(s[0], s[5], s[10], s[15]);
	Gfunc_v35(s[1], s[6], s[11], s[12]);
	Gfunc_v35(s[2], s[7], s[8],  s[13]);
	Gfunc_v35(s[3], s[4], s[9],  s[14]);
}

static __device__ __forceinline__ void round_lyra_v30(uint64_t *s)
{
	Gfunc_v30(s[0], s[4], s[8], s[12]);
	Gfunc_v30(s[1], s[5], s[9], s[13]);
	Gfunc_v30(s[2], s[6], s[10], s[14]);
	Gfunc_v30(s[3], s[7], s[11], s[15]);
	Gfunc_v30(s[0], s[5], s[10], s[15]);
	Gfunc_v30(s[1], s[6], s[11], s[12]);
	Gfunc_v30(s[2], s[7], s[8], s[13]);
	Gfunc_v30(s[3], s[4], s[9], s[14]);
}



__global__ void __launch_bounds__(256, 1) lyra2_gpu_hash_32_v30(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t state[16];
#pragma unroll
		for (int i = 0; i<4; i++) { state[i] = outputHash[threads*i + thread]; } //password
#pragma unroll
		for (int i = 0; i<4; i++) { state[i + 4] = state[i]; } //salt 
#pragma unroll
		for (int i = 0; i<8; i++) { state[i + 8] = devectorize(blake2b_IV[i]); }

		//     blake2blyra x2 
#pragma unroll 24
		for (int i = 0; i<24; i++) { round_lyra_v30(state); } //because 12 is not enough

		uint64_t Matrix[96][8]; // not cool
		/// reducedSqueezeRow0
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
int idx = 84-12*i;
#pragma unroll 12
			for (int j = 0; j<12; j++) { Matrix[j + idx][0] = state[j]; }
			round_lyra_v30(state);
		}

		/// reducedSqueezeRow1
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
int idx0= 12*i;
int idx1= 84-idx0; 
#pragma unroll 12
			for (int j = 0; j<12; j++) { state[j] ^= Matrix[j + idx0][0]; }
			round_lyra_v30(state);
#pragma unroll 12  
			for (int j = 0; j<12; j++) { Matrix[j + idx1][1] = Matrix[j + idx0][0] ^ state[j]; }
		}


		reduceDuplexRowSetup_v30(1, 0, 2);
		reduceDuplexRowSetup_v30(2, 1, 3);
		reduceDuplexRowSetup_v30(3, 0, 4);
		reduceDuplexRowSetup_v30(4, 3, 5);
		reduceDuplexRowSetup_v30(5, 2, 6);
		reduceDuplexRowSetup_v30(6, 1, 7);



		uint64_t rowa;
		rowa = state[0] & 7;
		reduceDuplexRow_v30(7, rowa, 0);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(0, rowa, 3);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(3, rowa, 6);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(6, rowa, 1);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(1, rowa, 4);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(4, rowa, 7);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(7, rowa, 2);
		rowa = state[0] & 7;
		reduceDuplexRow_v30(2, rowa, 5);

		absorbblock_v30(rowa);


#pragma unroll
		for (int i = 0; i<4; i++) {
			outputHash[threads*i + thread] = state[i];
		} //password


	} //thread
}


__global__ void __launch_bounds__(256, 1) lyra2_gpu_hash_32(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 state[16];
#pragma unroll
		for (int i = 0; i<4; i++) { LOHI(state[i].x, state[i].y, outputHash[threads*i + thread]); } //password
#pragma unroll
		for (int i = 0; i<4; i++) { state[i + 4] = state[i]; } //salt 
#pragma unroll
		for (int i = 0; i<8; i++) { state[i + 8] = blake2b_IV[i]; }

		//     blake2blyra x2 
#pragma unroll 24
		for (int i = 0; i<24; i++) { round_lyra_v35(state); } //because 12 is not enough

		uint2 Matrix[96][8]; // not cool

		/// reducedSqueezeRow0
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { Matrix[j + 84 - 12 * i][0] = state[j]; }
			round_lyra_v35(state);
		}

		/// reducedSqueezeRow1
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { state[j] ^= Matrix[j + 12 * i][0]; }
			round_lyra_v35(state);
#pragma unroll 12  
			for (int j = 0; j<12; j++) { Matrix[j + 84 - 12 * i][1] = Matrix[j + 12 * i][0] ^ state[j]; }
		}

		reduceDuplexRowSetup(1, 0, 2);
		reduceDuplexRowSetup(2, 1, 3);
		reduceDuplexRowSetup(3, 0, 4);
		reduceDuplexRowSetup(4, 3, 5);
		reduceDuplexRowSetup(5, 2, 6);
		reduceDuplexRowSetup(6, 1, 7);



		uint32_t rowa;
		rowa = state[0].x & 7;
		reduceDuplexRow(7, rowa, 0);
		rowa = state[0].x & 7;
		reduceDuplexRow(0, rowa, 3);
		rowa = state[0].x & 7;
		reduceDuplexRow(3, rowa, 6);
		rowa = state[0].x & 7;
		reduceDuplexRow(6, rowa, 1);
		rowa = state[0].x & 7;
		reduceDuplexRow(1, rowa, 4);
		rowa = state[0].x & 7;
		reduceDuplexRow(4, rowa, 7);
		rowa = state[0].x & 7;
		reduceDuplexRow(7, rowa, 2);
		rowa = state[0].x & 7;
		reduceDuplexRow(2, rowa, 5);

		absorbblock(rowa);


#pragma unroll
		for (int i = 0; i<4; i++) {
			outputHash[threads*i + thread] = devectorize(state[i]);
		} //password


	} //thread
}

__global__ void __launch_bounds__(256, 1) lyra2_gpu_hash_32_test(int threads, uint32_t startNounce, uint64_t *outputHash)
{

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint2 state[16];
#pragma unroll
		for (int i = 0; i<4; i++) { LOHI(state[i].x, state[i].y, outputHash[threads*i + thread]); } //password
#pragma unroll
		for (int i = 0; i<4; i++) { state[i + 4] = state[i]; } //salt 
#pragma unroll
		for (int i = 0; i<8; i++) { state[i + 8] = blake2b_IV[i]; }

		//     blake2blyra x2 
#pragma unroll 24
		for (int i = 0; i<24; i++) { round_lyra_v35(state); } //because 12 is not enough

		uint2 Matrix[12][8][8]; // not cool

		/// reducedSqueezeRow0
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { Matrix[j][7-i][0] = state[j]; }
			round_lyra_v35(state);
		}

		/// reducedSqueezeRow1
#pragma unroll 8 
		for (int i = 0; i < 8; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) { state[j] ^= Matrix[j][i][0]; }
			round_lyra_v35(state);
#pragma unroll 12  
			for (int j = 0; j<12; j++) { Matrix[j][7-i][1] = Matrix[j][i][0] ^ state[j]; }
		}

		reduceDuplexRowSetup_test(1, 0, 2);
		reduceDuplexRowSetup_test(2, 1, 3);
		reduceDuplexRowSetup_test(3, 0, 4);
		reduceDuplexRowSetup_test(4, 3, 5);
		reduceDuplexRowSetup_test(5, 2, 6);
		reduceDuplexRowSetup_test(6, 1, 7);



		uint64_t rowa;
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(7, rowa, 0);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(0, rowa, 3);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(3, rowa, 6);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(6, rowa, 1);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(1, rowa, 4);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(4, rowa, 7);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(7, rowa, 2);
		rowa = devectorize(state[0]) & 7;
		reduceDuplexRow_test(2, rowa, 5);

		absorbblock_test(rowa);


#pragma unroll
		for (int i = 0; i<4; i++) {
			outputHash[threads*i + thread] = devectorize(state[i]);
		} //password


	} //thread
}

   
void lyra2_cpu_init(int thr_id, int threads)
{
//not used    	
} 


__host__ void lyra2_cpu_hash_32(int thr_id, int threads, uint32_t startNounce, uint64_t *d_outputHash, int order)
{
	
	const int threadsperblock = 256;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;
	
	if (compute_version[thr_id]>=35) {
	lyra2_gpu_hash_32 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	}
	else {  // kernel for compute30 card
	lyra2_gpu_hash_32_v30 << <grid, block, shared_size >> >(threads, startNounce, d_outputHash);
	}
    
	MyStreamSynchronize(NULL, order, thr_id);

}

